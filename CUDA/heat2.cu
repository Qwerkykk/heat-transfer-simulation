
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include"/usr/local/cuda/include/cuda_runtime.h"
#define STEPS 5000

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void init(float *d_a, float *d_b, int x_dim, int y_dim) {
    int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int elements = (x_dim * y_dim);
    for (int i = thread_id; i < elements; i += stride) {
        int j_index = i % y_dim;
        int i_index = i / y_dim;
        //d_a[i] = (float) (i_index * (x_dim - i_index - 1) * j_index * (y_dim - j_index - 1));
        //d_b[i] = (float) d_a[i];
        //*(d_a + i_index * sd[1] + j_index) = (float) ((i_index + c[1]*sd[0]) * (d[0] - (i_index + c[1]*sd[0]) - 1) * (j_index + c[0]*sd[1]) * (d[1] - (j_index + c[0]*sd[1]) - 1));
        unsigned long int value = i_index * (x_dim - i_index - 1) * j_index * (y_dim - j_index - 1);
        *(d_a + i_index * y_dim + j_index) = (float)(value);
        *(d_b + i_index * y_dim + j_index) = (float)(value);
    }
    __syncthreads();
}

void prtdat(int nx, int ny, float *u1, char *fnam) {
    int ix, iy;
    FILE *fp;
    
    fp = fopen(fnam, "w");
    for (iy = ny-1; iy >= 0; iy--) {
      for (ix = 0; ix <= nx-1; ix++) {
        if (ix == nx-1)
          fprintf(fp, "%12.1f           \n", *(u1+ix*ny+iy));
        else
          fprintf(fp, "%12.1f            ", *(u1+ix*ny+iy));
        }
      }
    fclose(fp);
}

__global__ void update(float *d_a, float *d_b, int x_dim, int y_dim){

    int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int elements = (x_dim * y_dim);
    for (int i = thread_id; i < elements; i += stride) {
        int j_index = i % y_dim;
        int i_index = i / y_dim;
      
          
        if ((i_index == 0 || i_index == x_dim-1) || (j_index == 0 || j_index == y_dim-1)) continue;
    
        struct Parms {
          float cx;
          float cy;
        } parms = {0.1, 0.1};
    
        *(d_a + i_index * y_dim + j_index) = *(d_b + i_index * y_dim + j_index)  +
                          parms.cx * (*(d_b + (i_index +1) * y_dim + j_index) +
                          *(d_b + (i_index -1) * y_dim + j_index) -
                          2.0 * *(d_b + i_index * y_dim + j_index)) +
                          parms.cy * (*(d_b + i_index * y_dim + j_index + 1) +
                        *(d_b + i_index * y_dim + j_index - 1) -
                          2.0 * *(d_b + i_index * y_dim + j_index));
    }
   
  
    __syncthreads();
  
  }
    

int main(int argc, char const *argv[]) {
    hipEvent_t start, stop, stop2;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&stop2);
    if (argc != 4) {
      printf("error give the two dimensions of the array\n");
      return -1;
    }
  
    int x_dimension, y_dimension;
    x_dimension = atoi(argv[1]);
    y_dimension = atoi(argv[2]);
  
    float *d_a, *d_b, *a, *b;
    int bytes = x_dimension*y_dimension*sizeof(float);
    gpuErrchk( hipMalloc((void**)&d_a, bytes));
    gpuErrchk( hipMalloc((void**)&d_b, bytes));
    a = (float*) malloc(bytes);
    b = (float*) malloc(bytes);
    float *temp;
  
    gpuErrchk( hipMemset(d_a, 0, bytes));
    gpuErrchk( hipMemset(d_b, 0, bytes));

    int size = atoi(argv[3]);

    dim3 block_number(size);
    dim3 thread_number(32);


    init<<<block_number, thread_number>>>(d_a, d_b, x_dimension, y_dimension);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  

    gpuErrchk( hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost));
    prtdat(x_dimension, y_dimension, a, "initial.dat");
    gpuErrchk(  hipEventRecord(start));
    for (size_t i = 0; i < STEPS; i++) {

        update<<<block_number, thread_number>>>(d_a, d_b, x_dimension, y_dimension);

        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        temp = d_a;
        d_a = d_b;
        d_b = temp;

    }
    gpuErrchk(  hipEventRecord(stop));
    gpuErrchk( hipMemcpy(a, temp, bytes, hipMemcpyDeviceToHost));
    gpuErrchk( hipEventRecord(stop2));
    //cudaMemcpy(b, d_b, bytes, cudaMemcpyDeviceToHost);



    prtdat(x_dimension, y_dimension, a, "final.dat");
    //prtdat(x_dimension, y_dimension, b, "2eend_res.dat");

    gpuErrchk( hipFree(d_a));
    gpuErrchk( hipFree(d_b));


    gpuErrchk(  hipEventSynchronize(stop));
    float milliseconds = 0;
    gpuErrchk( hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time: %f\n", milliseconds);

    gpuErrchk( hipEventElapsedTime(&milliseconds, start, stop2));
    printf("Time with transfer: %f\n", milliseconds);
    return 0;

}